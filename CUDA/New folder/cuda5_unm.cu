#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage wit unified memory.
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "pic_type.h"


__global__ void kernel_flag (CudaPic t_color_pic )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_pic.m_size.y ) return;
	if ( l_x >= t_color_pic.m_size.x ) return;
	/*
	int r = 50;
	int sx = 150 - l_x;
	int sy = 75 - l_y;

	uchar3 l_bgr = { 255, 255, 255 };

	if (sx * sx + sy*sy >= r*r -70 && sx * sx + sy*sy <= r*r + 70)
		l_bgr = { 0, 0, 0 };
	*/
	uchar3 l_bgr = { 255, 255, 255 };
	if (l_x < 125){
		l_bgr = { 255, 0, 0 };

	}
	if (l_x > 250)
		l_bgr = { 0, 0, 255 };

	// Store point into image
	t_color_pic.m_p_uchar3[ l_y * t_color_pic.m_size.x + l_x ] = l_bgr;
}

void cu_create_flag(CudaPic t_color_pic)
{
	hipError_t l_cerr;
	dim3 blocks (35,20);
	dim3 threads (10,10);
	kernel_flag<<<blocks, threads>>>( t_color_pic );
	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

__global__ void kernel_rotate(CudaPic t_color_pic, CudaPic t_color_pic_rotated)
{
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_pic.m_size.y ) return;
	if ( l_x >= t_color_pic.m_size.x ) return;

	int newX = t_color_pic_rotated.m_size.x - l_y;
	int newY = l_x;

	t_color_pic_rotated.m_p_uchar3[ newY * t_color_pic_rotated.m_size.x +  newX] = t_color_pic.m_p_uchar3[ l_y * t_color_pic.m_size.x + l_x ];
}

void cu_rotate(CudaPic t_color_pic, CudaPic t_color_pic_rotated)
{
	hipError_t l_cerr;
	dim3 blocks (35,20);
	dim3 threads (10,10);
	kernel_rotate<<<blocks, threads>>>( t_color_pic, t_color_pic_rotated );
	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------


__global__ void kernel_insert(CudaPic t_big_pic, CudaPic t_small_pic, int2 t_position)
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_small_pic.m_size.y ) return;
	if ( l_x >= t_small_pic.m_size.x ) return;
	int l_by = l_y + t_position.y;
	int l_bx = l_x + t_position.x;
	if ( l_by >= t_big_pic.m_size.y || l_by < 0 ) return;
	if ( l_bx >= t_big_pic.m_size.x || l_bx < 0 ) return;

	t_big_pic.m_p_uchar3[ l_by * t_big_pic.m_size.x + l_bx ] = t_small_pic.m_p_uchar3[ l_y * t_small_pic.m_size.x + l_x ];
}

void cu_insert( CudaPic t_big_pic, CudaPic t_small_pic, int2 t_position )
{
	hipError_t l_cerr;
	dim3 blocks (t_small_pic.m_size.x/10, t_small_pic.m_size.y/10);
	dim3 threads (10,10);
	kernel_insert<<<blocks, threads>>>( t_big_pic, t_small_pic, t_position );
	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

__global__ void kernel_wave(CudaPic old_pic, CudaPic new_pic)
{
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= old_pic.m_size.y ) return;
	if ( l_x >= old_pic.m_size.x ) return;

	float a = l_x;
	int l_ny = l_y +50 - sin(a/20)*20;
	int l_nx = l_x;
	int colourscale = (1 - sin(a/20))*70;
	if ( l_ny >= new_pic.m_size.y || l_ny < 0 ) return;
	if ( l_nx >= new_pic.m_size.x || l_nx < 0 ) return;

	new_pic.m_p_uchar3[ l_ny * new_pic.m_size.x + l_nx ] = old_pic.m_p_uchar3[ l_y * old_pic.m_size.x + l_x ];
	if(new_pic.m_p_uchar3[ l_ny * new_pic.m_size.x + l_nx ].x > colourscale)
		new_pic.m_p_uchar3[ l_ny * new_pic.m_size.x + l_nx ].x -= colourscale;
	if(new_pic.m_p_uchar3[ l_ny * new_pic.m_size.x + l_nx ].y > colourscale)
		new_pic.m_p_uchar3[ l_ny * new_pic.m_size.x + l_nx ].y -= colourscale;
	if(new_pic.m_p_uchar3[ l_ny * new_pic.m_size.x + l_nx ].z > colourscale)
		new_pic.m_p_uchar3[ l_ny * new_pic.m_size.x + l_nx ].z -= colourscale;

}

void cu_wave(CudaPic old_pic, CudaPic new_pic)
{
	hipError_t l_cerr;
	dim3 blocks (old_pic.m_size.x/10, old_pic.m_size.y/10);
	dim3 threads (10,10);
	kernel_wave<<<blocks, threads>>>( old_pic, new_pic);
	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );
	hipDeviceSynchronize();
}

__global__ void kernel_cut(CudaPic t_orig_pic, CudaPic t_cut_pic, int2 t_position)
{
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_cut_pic.m_size.y ) return;
	if ( l_x >= t_cut_pic.m_size.x ) return;
	int l_oy = l_y + t_position.y;
	int l_ox = l_x + t_position.x;
	if ( l_oy >= t_orig_pic.m_size.y || l_oy < 0 ) return;
	if ( l_ox >= t_orig_pic.m_size.x || l_ox < 0 ) return;

	t_cut_pic.m_p_uchar3[ l_y * t_cut_pic.m_size.x + l_x ] = t_orig_pic.m_p_uchar3[ l_oy * t_orig_pic.m_size.x + l_ox ];
}


void cu_cut(CudaPic t_orig_pic, CudaPic t_cut_pic, int2 t_position)
{
	hipError_t l_cerr;
	dim3 blocks (t_cut_pic.m_size.x/10, t_cut_pic.m_size.y/10);
	dim3 threads (10,10);
	kernel_cut<<<blocks, threads>>>( t_orig_pic, t_cut_pic, t_position );
	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

__global__ void resize(CudaPic t_orig_pic, CudaPic t_resized_pic)
{

}

void cu_resize(CudaPic t_orig_pic, CudaPic t_resized_pic)
{

}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_insertimage( CudaPic t_big_pic, CudaPic t_small_pic, int2 t_position )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_small_pic.m_size.y ) return;
	if ( l_x >= t_small_pic.m_size.x ) return;
	int l_by = l_y + t_position.y;
	int l_bx = l_x + t_position.x;
	if ( l_by >= t_big_pic.m_size.y || l_by < 0 ) return;
	if ( l_bx >= t_big_pic.m_size.x || l_bx < 0 ) return;

	// Get point from small image
	uchar4 l_fg_bgra = t_small_pic.m_p_uchar4[ l_y * t_small_pic.m_size.x + l_x ];
	uchar3 l_bg_bgr = t_big_pic.m_p_uchar3[ l_by * t_big_pic.m_size.x + l_bx ];
	uchar3 l_bgr = { 0, 0, 0 };

	// compose point from small and big image according alpha channel
	l_bgr.x = l_fg_bgra.x * l_fg_bgra.w / 255 + l_bg_bgr.x * ( 255 - l_fg_bgra.w ) / 255;
	l_bgr.y = l_fg_bgra.y * l_fg_bgra.w / 255 + l_bg_bgr.y * ( 255 - l_fg_bgra.w ) / 255;
	l_bgr.z = l_fg_bgra.z * l_fg_bgra.w / 255 + l_bg_bgr.z * ( 255 - l_fg_bgra.w ) / 255;

	// Store point into image
	t_big_pic.m_p_uchar3[ l_by * t_big_pic.m_size.x + l_bx ] = l_bgr;
}

void cu_insertimage( CudaPic t_big_pic, CudaPic t_small_pic, int2 t_position )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 32;
	dim3 l_blocks( ( t_small_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( t_small_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_insertimage<<< l_blocks, l_threads >>>( t_big_pic, t_small_pic, t_position );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

// Demo kernel to create chess board
__global__ void kernel_chessboard( CudaPic t_color_pic )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_pic.m_size.y ) return;
	if ( l_x >= t_color_pic.m_size.x ) return;

	unsigned char b_or_w = 255 * ( ( blockIdx.x + blockIdx.y ) & 1 );

	// Store point into image
	t_color_pic.m_p_uchar3[ l_y * t_color_pic.m_size.x + l_x ] = { b_or_w, b_or_w, b_or_w };
}

void cu_create_chessboard( CudaPic t_color_pic, int t_square_size )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	dim3 l_blocks( ( t_color_pic.m_size.x + t_square_size - 1 ) / t_square_size,
			       ( t_color_pic.m_size.y + t_square_size - 1 ) / t_square_size );
	dim3 l_threads( t_square_size, t_square_size );
	kernel_chessboard<<< l_blocks, l_threads >>>( t_color_pic );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_alphaimg( CudaPic t_color_pic, uchar3 t_color )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_pic.m_size.y ) return;
	if ( l_x >= t_color_pic.m_size.x ) return;

	int l_diagonal = sqrtf( t_color_pic.m_size.x * t_color_pic.m_size.x + t_color_pic.m_size.y * t_color_pic.m_size.y );
	int l_dx = l_x - t_color_pic.m_size.x / 2;
	int l_dy = l_y - t_color_pic.m_size.y / 2;
	int l_dxy = sqrtf( l_dx * l_dx + l_dy * l_dy ) - l_diagonal / 2;

	// Store point into image
	t_color_pic.m_p_uchar4[ l_y * t_color_pic.m_size.x + l_x ] =
		{ t_color.x, t_color.y, t_color.z, ( unsigned char ) ( 255 - 255 * l_dxy / ( l_diagonal / 2 ) ) };
}

void cu_create_alphaimg( CudaPic t_color_pic, uchar3 t_color )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 32;
	dim3 l_blocks( ( t_color_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( t_color_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_alphaimg<<< l_blocks, l_threads >>>( t_color_pic, t_color );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}
