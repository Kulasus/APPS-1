#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage without unified memory.
//
// Simple animation.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "pic_type.h"
#include "animation.h"

// Demo kernel to create chess board
__global__ void kernel_creategradient( CudaPic t_color_pic )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_pic.m_size.y ) return;
	if ( l_x >= t_color_pic.m_size.x ) return;

	int l_dy = l_x * t_color_pic.m_size.y / t_color_pic.m_size.x + l_y - t_color_pic.m_size.y;
	unsigned char l_color = 255 * abs( l_dy ) / t_color_pic.m_size.y;

	uchar3 l_bgr = ( l_dy < 0 ) ? ( uchar3 ) { l_color, 255 - l_color, 0 } : ( uchar3 ) { 0, 255 - l_color, l_color };

	// Store point into image
	t_color_pic.m_p_uchar3[ l_y * t_color_pic.m_size.x + l_x ] = l_bgr;
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_insertimage( CudaPic t_big_pic, CudaPic t_small_pic, int2 t_position )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_small_pic.m_size.y ) return;
	if ( l_x >= t_small_pic.m_size.x ) return;
	int l_by = l_y + t_position.y;
	int l_bx = l_x + t_position.x;
	if ( l_by >= t_big_pic.m_size.y || l_by < 0 ) return;
	if ( l_bx >= t_big_pic.m_size.x || l_bx < 0 ) return;

	// Store point into image
	t_big_pic.at3(l_by, l_bx) = t_small_pic.at3(l_y, l_x);
}

void cu_insertimage( CudaPic t_big_pic, CudaPic t_small_pic, int2 t_position )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 32;
	dim3 l_blocks( ( t_small_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( t_small_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_insertimage<<< l_blocks, l_threads >>>( t_big_pic, t_small_pic, t_position );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

void Animation::start( CudaPic t_bg_pic, CudaPic t_ins_pic )
{
	if ( m_initialized ) return;
	hipError_t l_cerr;

	m_cuda_bg_pic = t_bg_pic;
	m_cuda_res_pic = t_bg_pic;
	m_cuda_ins_pic = t_ins_pic;

	// Memory allocation in GPU device
	// Memory for background
	l_cerr = hipMalloc( &m_cuda_bg_pic.m_p_void, m_cuda_bg_pic.m_size.x * m_cuda_bg_pic.m_size.y * sizeof( uchar3 ) );
	if ( l_cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	// Creation of background gradient
	int l_block_size = 32;
	dim3 l_blocks( ( m_cuda_bg_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( m_cuda_bg_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_creategradient<<< l_blocks, l_threads >>>( m_cuda_bg_pic );

	m_initialized = 1;
}

void Animation::next( CudaPic t_res_pic, int2 t_position )
{
	if ( !m_initialized ) return;

	hipError_t cerr;

	// Copy data internally GPU from background into result
	cerr = hipMemcpy( m_cuda_res_pic.m_p_void, m_cuda_bg_pic.m_p_void, m_cuda_bg_pic.m_size.x * m_cuda_bg_pic.m_size.y * sizeof( uchar3 ), hipMemcpyDeviceToDevice );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// insert picture
	int l_block_size = 32;
	dim3 l_blocks( ( m_cuda_ins_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( m_cuda_ins_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_insertimage<<< l_blocks, l_threads >>>( m_cuda_res_pic, m_cuda_ins_pic, t_position );

	// Copy data to GPU device
	cerr = hipMemcpy( t_res_pic.m_p_void, m_cuda_res_pic.m_p_void, m_cuda_res_pic.m_size.x * m_cuda_res_pic.m_size.y * sizeof( uchar3 ), hipMemcpyDeviceToHost );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

}

void Animation::stop()
{
	if ( !m_initialized ) return;

	hipFree( m_cuda_bg_pic.m_p_void );
	hipFree( m_cuda_res_pic.m_p_void );
	hipFree( m_cuda_ins_pic.m_p_void );

	m_initialized = 0;
}

// -----------------------------------------------------------------------------------------------

__global__ void kernel_scale(CudaPic t_orig, CudaPic t_resize)
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if (l_y >= t_resize.m_size.y) return;
	if (l_x >= t_resize.m_size.x) return;

	float l_scale_x = t_orig.m_size.x - 1;
	float l_scale_y = t_orig.m_size.y - 1;
	l_scale_x /= t_resize.m_size.x;
	l_scale_y /= t_resize.m_size.y;

	int l_resize_x = l_x;
	int l_resize_y = l_y;
	// new real position
	float l_orig_x = l_resize_x * l_scale_x;
	float l_orig_y = l_resize_y * l_scale_y;
	// diff x and y
	float l_diff_x = l_orig_x - (int)l_orig_x;
	float l_diff_y = l_orig_y - (int)l_orig_y;

	uchar3 px00 = t_orig.at3((int)l_orig_y, (int)l_orig_x);
	uchar3 px01 = t_orig.at3((int)l_orig_y, 1 + (int)l_orig_x);
	uchar3 px10 = t_orig.at3(1 + (int)l_orig_y, (int)l_orig_x );
	uchar3 px11 = t_orig.at3(1 + (int)l_orig_y, 1 + (int)l_orig_x);

	uchar bgr00[3] = {px00.x, px00.y, px00.z};
	uchar bgr01[3] = {px01.x, px01.y, px01.z};
	uchar bgr10[3] = {px10.x, px10.y, px10.z};
	uchar bgr11[3] = {px11.x, px11.y, px11.z};

	uchar bgr[3];

	for ( int i = 0; i < 3; i++ )
	{
		// color calculation
		bgr[i] = bgr00[i] * (1 - l_diff_y) * (1 - l_diff_x) +
				   bgr01[i] * (1 - l_diff_y) * (l_diff_x) +
				   bgr10[i] * (l_diff_y) * (1 - l_diff_x) +
				   bgr11[i] * (l_diff_y) * (l_diff_x);
	}
	t_resize.at3(l_y, l_x) = {bgr[0], bgr[1], bgr[2]};
}

void cu_scale(CudaPic original, CudaPic resize)
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 32;
	dim3 l_blocks( ( resize.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( resize.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_scale<<< l_blocks, l_threads >>>(original, resize);

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}






















